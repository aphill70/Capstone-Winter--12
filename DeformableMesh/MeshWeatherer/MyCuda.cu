#include "hip/hip_runtime.h"
#include "MyCuda.h"
#include "Pair.h"

#include <cstdio>


static void HandleError( hipError_t err, const char * file, int line ) {

	if ( err != hipSuccess ) {

		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__global__ void relabelKernel( const Pair * pairs, int * results, int pairsSize, int resultsSize ) {
	
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	while ( tid < pairsSize ) {

		if ( pairs[ tid ].tetrahedron.contains( pairs[ tid ].circumcenter.point ) ) {

			results[ pairs[ tid ].circumcenter.index ] = pairs[ tid ].tetrahedron.type;
		}

		tid += stride;
	}
}


void relabel( const Pair * pairs, int * results, int pairsSize, int resultsSize ) {

	int * dev_results;
	Pair * dev_pairs;
	
	HANDLE_ERROR( hipMalloc( ( int ** ) &dev_results, resultsSize * sizeof( int ) ) );
	HANDLE_ERROR( hipMemcpy( dev_results, results, resultsSize * sizeof( int ), hipMemcpyHostToDevice ) );

	HANDLE_ERROR( hipMalloc( ( Pair ** ) &dev_pairs, pairsSize * sizeof( Pair ) ) );
	HANDLE_ERROR( hipMemcpy( dev_pairs, pairs, pairsSize * sizeof( Pair ), hipMemcpyHostToDevice ) );

	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0 );
	relabelKernel<<<1024,512>>>( dev_pairs, dev_results, pairsSize, resultsSize );
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	float elapsedTime;
	HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );
	printf( "Time to run kernel: %.4f sec\n", elapsedTime / 1000.0f );

	HANDLE_ERROR( hipMemcpy( results, dev_results, resultsSize * sizeof( int ), hipMemcpyDeviceToHost ) );

	HANDLE_ERROR( hipFree( dev_results ) );
	HANDLE_ERROR( hipFree( dev_pairs ) );
	HANDLE_ERROR( hipEventDestroy( start ) );
	HANDLE_ERROR( hipEventDestroy( stop ) );
}
